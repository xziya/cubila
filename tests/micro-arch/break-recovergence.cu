#include <hip/hip_runtime.h>

#include <cstdint>
#include <cstdio>

__global__ void break_recovergence_test() {
  __shared__ volatile int32_t shared_var;

  const int32_t tid = threadIdx.x;

  if (tid == 0) { shared_var = 0; }

  while (shared_var != tid)
    ;
  shared_var++;

  return;
}

int main() {
  printf("start to run test...\n");

  break_recovergence_test<<<1, 32>>>();
  hipDeviceSynchronize();

  printf("end...\n");

  return 0;
}
